#include "hip/hip_runtime.h"
#include "nicpcuda.cuh"
#include "functors.cuh"
#include "nicputils.h"
#include "nicpslamargs.cuh"

using namespace CUDACommon;
using namespace Common;

namespace
{
	void PrepareMatricesForParallelSVD(const GpuCloud& cloudBefore, const GpuCloud& cloudAfter, int batchSize, NonIterativeSLAMArgs& args)
	{
		int cloudSize = std::min(cloudBefore.size(), cloudAfter.size());

		for (int i = 0; i < batchSize; i++)
		{
			// Generate permutation
			std::vector<int> h_permutation = GetRandomPermutationVector(cloudSize);
			IndexIterator d_permutation(h_permutation.size());
			thrust::copy(h_permutation.begin(), h_permutation.end(), d_permutation.begin());
			ApplyPermutation(args.alignedCloudBefore, d_permutation, args.permutedCloudBefore);
			ApplyPermutation(args.alignedCloudAfter, d_permutation, args.permutedCloudAfter);

			// Create counting iterators
			auto beforeCountingBegin = thrust::make_counting_iterator<int>(0);
			auto beforeCountingEnd = thrust::make_counting_iterator<int>(args.permutedCloudBefore.size());
			auto afterCountingBegin = thrust::make_counting_iterator<int>(0);
			auto afterCountingEnd = thrust::make_counting_iterator<int>(args.permutedCloudAfter.size());

			// Create array for SVD
			const auto beforeZipBegin = thrust::make_zip_iterator(thrust::make_tuple(beforeCountingBegin, args.permutedCloudBefore.begin()));
			const auto beforeZipEnd = thrust::make_zip_iterator(thrust::make_tuple(beforeCountingEnd, args.permutedCloudBefore.end()));
			auto convertBefore = Functors::GlmToCuBlas(true, args.permutedCloudBefore.size(), args.preparedBeforeClouds[i]);
			thrust::for_each(thrust::device, beforeZipBegin, beforeZipEnd, convertBefore);
			const auto afterZipBegin = thrust::make_zip_iterator(thrust::make_tuple(afterCountingBegin, args.permutedCloudAfter.begin()));
			const auto afterZipEnd = thrust::make_zip_iterator(thrust::make_tuple(afterCountingEnd, args.permutedCloudAfter.end()));
			auto convertAfter = Functors::GlmToCuBlas(true, args.permutedCloudAfter.size(), args.preparedAfterClouds[i]);
			thrust::for_each(thrust::device, afterZipBegin, afterZipEnd, convertAfter);
		}
	}

	void GetSVDResultParallel(const GpuCloud& cloudBefore, const GpuCloud& cloudAfter, int batchSize, NonIterativeSLAMArgs& args, thrust::host_vector<glm::mat3>& outputBefore, thrust::host_vector<glm::mat3>& outputAfter)
	{
		PrepareMatricesForParallelSVD(cloudBefore, cloudAfter, batchSize, args);

		// Run SVD for cloud before
		args.svdHelperBefore.RunSVD(args.preparedBeforeClouds, batchSize);
		outputBefore = args.svdHelperBefore.GetHostMatricesVT();

		args.svdHelperAfter.RunSVD(args.preparedAfterClouds, batchSize);
		outputAfter = args.svdHelperAfter.GetHostMatricesVT();
	}

	void GetSubcloud(const GpuCloud& cloud, int subcloudSize, GpuCloud& outputSubcloud)
	{
		if (subcloudSize >= cloud.size())
			outputSubcloud = cloud;

		outputSubcloud.resize(subcloudSize);

		std::vector<int> h_indices = GetRandomPermutationVector(cloud.size());
		h_indices.resize(subcloudSize);
		thrust::device_vector<int> d_indices(h_indices);

		auto permutationIterBegin = thrust::make_permutation_iterator(cloud.begin(), d_indices.begin());
		auto permutationIterEnd = thrust::make_permutation_iterator(cloud.end(), d_indices.end());
		thrust::copy(permutationIterBegin, permutationIterEnd, outputSubcloud.begin());
	}
}

std::pair<glm::mat3, glm::vec3> CudaNonIterative(const GpuCloud& before, const GpuCloud& after, int* repetitions, float* error, float eps, int maxRepetitions, int batchSize, ApproximationType approximationType, const int subcloudSize)
{
	// Set the number of results to store - 1 for Full, 5 for Hybrid, unused for None
	auto resultsNumber = approximationType == ApproximationType::Full ? 1 : 5;
	
	// Prepare stuctures for storing transformation results for subsequent executions
	std::pair<glm::mat3, glm::vec3> bestTransformation;
	std::pair<glm::mat3, glm::vec3> currentTransformation;
	thrust::host_vector<glm::mat3> matricesBefore(batchSize);
	thrust::host_vector<glm::mat3> matricesAfter(batchSize);
	std::vector<NonIterativeSlamResult> bestResults(resultsNumber);

	// Split number of repetitions to batches
	auto batchesCount = maxRepetitions / batchSize;
	auto lastBatchSize = maxRepetitions % batchSize;
	auto threadsToRun = batchSize;

	// Prepare helper structures
	auto minError = *error = std::numeric_limits<float>::max();
	GpuCloud subcloud(subcloudSize);
	GpuCloud transformedSubcloud(subcloudSize);
	GetSubcloud(before, subcloudSize, subcloud);
	thrust::device_vector<int> permutedIndices(subcloudSize);
	thrust::device_vector<int> nonPermutedIndices(before.size());
	thrust::counting_iterator<int> helperIterator(0);
	thrust::copy(helperIterator, helperIterator + before.size(), nonPermutedIndices.begin());

	NonIterativeSLAMArgs args(batchSize, before, after);

	auto centroidBefore = CalculateCentroid(before);
	auto centroidAfter = CalculateCentroid(after);

	*repetitions = 0;

	// Run actual SLAM in batches
	for (int i = 0; i <= batchesCount; i++)
	{
		if (i == batchesCount)
		{
			if (lastBatchSize != 0)
				threadsToRun = lastBatchSize;
			else
				break;
		}

		GetSVDResultParallel(before, after, threadsToRun, args, matricesBefore, matricesAfter);
		*repetitions += threadsToRun;

		for (int j = 0; j < threadsToRun; j++)
		{
			glm::mat3 rotationMatrix = matricesAfter[j] * glm::transpose(matricesBefore[j]);
			glm::vec3 translationVector = centroidAfter - (rotationMatrix * centroidBefore);
			currentTransformation = std::make_pair(rotationMatrix, translationVector);

			// If using approximation, get error without finding correspondences - quick and efficient for poorly permuted clouds
			// Do find correspondences if using approximationType == None
			if (approximationType == ApproximationType::None)
			{
				TransformCloud(subcloud, transformedSubcloud, ConvertToTransformationMatrix(currentTransformation.first, currentTransformation.second));
				GetCorrespondingPoints(permutedIndices, transformedSubcloud, after);
				*error = GetMeanSquaredError(permutedIndices, transformedSubcloud, after);

				if (*error < minError)
				{
					minError = *error;
					bestTransformation = currentTransformation;

					if (minError <= eps)
					{
						printf("Error: %f\n", minError);
						args.Free();
						return currentTransformation;
					}
				}
			}
			else
			{
				*error = GetMeanSquaredError(nonPermutedIndices, before, after);

				NonIterativeSlamResult transformationResult(rotationMatrix, translationVector, *error);
				StoreResultIfOptimal(bestResults, transformationResult, resultsNumber);
			}
		}
	}

	// If using hybrid approximation, select best result
	// If using full approximation, calculate exact error for the best result
	if (approximationType != ApproximationType::None)
	{
		minError = std::numeric_limits<float>::max();
		for (int i = 0; i < bestResults.size(); i++)
		{
			TransformCloud(subcloud, transformedSubcloud, bestResults[i].getTransformationMatrix());
			GetCorrespondingPoints(permutedIndices, transformedSubcloud, after);
			*error = GetMeanSquaredError(permutedIndices, transformedSubcloud, after);

			if (*error < minError)
			{
				minError = *error;
				bestTransformation = bestResults[i].getTransformation();

				if (minError <= eps)
				{
					args.Free();
					return bestTransformation;
				}
			}
		}
	}

	*error = minError;
	args.Free();
	return bestTransformation;
}

std::pair<glm::mat3, glm::vec3> GetCudaNicpTransformationMatrix(
	const std::vector<Point_f>& before,
	const std::vector<Point_f>& after,
	float eps,
	int maxRepetitions,
	int batchSize,
	Common::ApproximationType approximationType,
	const int subcloudSize,
	int* repetitions,
	float* error)
{
	GpuCloud gpuBefore(before.size());
	GpuCloud gpuAfter(after.size());

	checkCudaErrors(hipMemcpy(thrust::raw_pointer_cast(gpuBefore.data()), before.data(), before.size() * sizeof(glm::vec3), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(thrust::raw_pointer_cast(gpuAfter.data()), after.data(), after.size() * sizeof(glm::vec3), hipMemcpyHostToDevice));

	const auto result = CudaNonIterative(gpuBefore, gpuAfter, repetitions, error, eps, maxRepetitions, batchSize, approximationType, subcloudSize);

	GpuCloud transformedCloud(gpuBefore.size());
	TransformCloud(gpuBefore, transformedCloud, ConvertToTransformationMatrix(result.first, result.second));
	thrust::device_vector<int> indices(gpuBefore.size());
	GetCorrespondingPoints(indices, transformedCloud, gpuAfter);
	*error = GetMeanSquaredError(indices, transformedCloud, gpuAfter);

	return result;
}

