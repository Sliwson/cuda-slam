#include "svdparams.cuh"

CudaSvdParams::CudaSvdParams(int beforeLength, int afterLength, int m, int n, bool useMatrixU, bool useMatrixV)
	:m(m), n(n), useMatrixU(useMatrixU), useMatrixV(useMatrixV)
{
	hipMalloc(&workBefore, beforeLength * n * sizeof(float));
	hipMalloc(&workAfter, afterLength * n * sizeof(float));
	hipMalloc(&multiplyResult, n * m * sizeof(float));
	hipblasCreate(&multiplyHandle);

	hipMalloc(&devInfo, sizeof(int));
	hipMalloc(&S, n * n * sizeof(float));
	if (useMatrixV)
		hipMalloc(&VT, n * n * sizeof(float));
	if (useMatrixU)
		hipMalloc(&U, m * m * sizeof(float));
	hipsolverDnCreate(&solverHandle);

	hipsolverDnSgesvd_bufferSize(solverHandle, m, n, &workSize);

	hipMalloc(&work, workSize * sizeof(float));
}

void CudaSvdParams::Free()
{
	hipFree(workBefore);
	hipFree(workAfter);
	hipFree(multiplyResult);
	hipblasDestroy(multiplyHandle);

	hipFree(work);
	hipFree(devInfo);
	hipFree(S);
	if (useMatrixV)
		hipFree(VT);
	if (useMatrixU)
		hipFree(U);
	hipsolverDnDestroy(solverHandle);
}
