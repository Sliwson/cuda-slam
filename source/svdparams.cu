#include "svdparams.cuh"

using namespace CUDACommon;

CudaSvdParams::CudaSvdParams(int beforeLength, int afterLength, int m, int n, bool useMatrixU, bool useMatrixV)
	:m(m), n(n), useMatrixU(useMatrixU), useMatrixV(useMatrixV)
{
	checkCudaErrors(hipMalloc((void**)&workBefore, beforeLength * n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&workAfter, afterLength * n * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&multiplyResult, n * m * sizeof(float)));
	hipblasCreate(&multiplyHandle);

	checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&S, n * n * sizeof(float)));
	if (useMatrixV)
		checkCudaErrors(hipMalloc((void**)&VT, n * n * sizeof(float)));
	if (useMatrixU)
		checkCudaErrors(hipMalloc((void**)&U, m * m * sizeof(float)));
	cusolveSafeCall(hipsolverDnCreate(&solverHandle));

	cusolveSafeCall(hipsolverDnSgesvd_bufferSize(solverHandle, m, n, &workSize));

	checkCudaErrors(hipMalloc((void**)&work, workSize * sizeof(float)));
}

void CudaSvdParams::Free()
{
	checkCudaErrors(hipFree(workBefore));
	checkCudaErrors(hipFree(workAfter));
	checkCudaErrors(hipFree(multiplyResult));
	hipblasDestroy(multiplyHandle);

	checkCudaErrors(hipFree(work));
	checkCudaErrors(hipFree(devInfo));
	checkCudaErrors(hipFree(S));
	if (useMatrixV)
		checkCudaErrors(hipFree(VT));
	if (useMatrixU)
		checkCudaErrors(hipFree(U));
	cusolveSafeCall(hipsolverDnDestroy(solverHandle));
}
