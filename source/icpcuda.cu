#include "hip/hip_runtime.h"
#include "icpcuda.cuh"
#include "functors.cuh"
#include "svdparams.cuh"

using namespace Common;
using namespace CUDACommon;

namespace
{
	__global__ void FindCorrespondences(int* result, const glm::vec3* before, const glm::vec3* after, int beforeSize, int afterSize)
	{
		int targetIdx = blockDim.x * blockIdx.x + threadIdx.x;
		if (targetIdx < beforeSize)
		{
			const glm::vec3 vector = before[targetIdx];
			int nearestIdx = 0;
			float smallestError = GetDistanceSquared(vector, after[0]);
			for (int i = 1; i < afterSize; i++)
			{
				const auto dist = GetDistanceSquared(vector, after[i]);
				if (dist < smallestError)
				{
					smallestError = dist;
					nearestIdx = i;
				}
			}

			result[targetIdx] = nearestIdx;
		}
	}

	void GetCorrespondingPoints(thrust::device_vector<int>& indices, const Cloud& before, const Cloud& after)
	{
#ifdef USE_CORRESPONDENCES_KERNEL
		int* dIndices = thrust::raw_pointer_cast(indices.data());
		const glm::vec3* dBefore = thrust::raw_pointer_cast(before.data());
		const glm::vec3* dAfter = thrust::raw_pointer_cast(after.data());
		int beforeSize = before.size();
		int afterSize = after.size();

		constexpr int threadsPerBlock = 256;
		const int blocksPerGrid = (beforeSize + threadsPerBlock - 1) / threadsPerBlock;
		FindCorrespondences << <blocksPerGrid, threadsPerBlock >> > (dIndices, dBefore, dAfter, beforeSize, afterSize);
		hipDeviceSynchronize();
#else
		const auto nearestFunctor = Functors::FindNearestIndex(after);
		thrust::transform(thrust::device, before.begin(), before.end(), indices.begin(), nearestFunctor);
#endif
	}

	glm::mat4 CudaICP(const Cloud& before, const Cloud& after)
	{
		const int maxIterations = 60;
		const float TEST_EPS = 1e-5;
		float previousError = std::numeric_limits<float>::max();

		int iterations = 0;
		glm::mat4 transformationMatrix(1.0f);
		glm::mat4 previousTransformationMatrix = transformationMatrix;

		//do not change before vector - copy it for calculations
		const int size = std::max(before.size(), after.size());
		Cloud workingBefore(size);
		Cloud alignBefore(size);
		Cloud alignAfter(size);
		thrust::device_vector<int> indices(before.size());
		thrust::copy(thrust::device, before.begin(), before.end(), workingBefore.begin());

		//allocate memory for cuBLAS
		CudaSvdParams params(size, size);

		while (iterations < maxIterations)
		{
			GetCorrespondingPoints(indices, workingBefore, after);

			transformationMatrix = LeastSquaresSVD(indices, workingBefore, after, alignBefore, alignAfter, params) * transformationMatrix;

			TransformCloud(before, workingBefore, transformationMatrix);
			float error = GetMeanSquaredError(indices, workingBefore, after);
			printf("Iteration: %d, error: %f\n", iterations, error);
			if (error < TEST_EPS)
				break;

			if (error > previousError)
			{
				printf("Error has increased, aborting\n");
				transformationMatrix = previousTransformationMatrix;
				break;
			}

			previousTransformationMatrix = transformationMatrix;
			previousError = error;
			iterations++;
		}

		params.Free();
		return transformationMatrix;
	}

	void CorrespondencesTest()
	{
		const int size = 100;
		thrust::device_vector<glm::vec3> input(size);
		thrust::device_vector<glm::vec3> output(size);
		thrust::device_vector<int> result(size);

		for (int i = 0; i < size; i++)
		{
			const auto vec = glm::vec3(i);
			input[i] = vec;
			output[size - i - 1] = vec;
		}

		GetCorrespondingPoints(result, input, output);
		thrust::host_vector<int> copy = result;
		bool ok = true;
		int hostArray[size];
		for (int i = 0; i < size; i++)
		{
			hostArray[i] = copy[i];
			if (copy[i] != size - i - 1)
				ok = false;
		}

		printf("Correspondence test [%s]\n", ok ? "OK" : "FAILED");
	}

	void MultiplicationTest()
	{
		const int size = 100;

		float ones[3 * size];
		for (int i = 0; i < 3 * size; i++)
			ones[i] = 1.f;

		CudaSvdParams params(size, size);
		hipMemcpy(params.workBefore, ones, 3 * size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(params.workAfter, ones, 3 * size * sizeof(float), hipMemcpyHostToDevice);

		CuBlasMultiply(params.workBefore, params.workAfter, params.multiplyResult, size, params);

		float result[9];
		hipMemcpy(result, params.multiplyResult, 9 * sizeof(float), hipMemcpyDeviceToHost);

		bool ok = true;
		for (int i = 0; i < 9; i++)
			if (abs(result[i] - size) > 1e-5)
				ok = false;

		printf("Multiplication test [%s]\n", ok ? "OK" : "FAILED");
		params.Free();
	}
}

void CudaTest()
{
	/****************************/
	//TESTS
	/****************************/
	CorrespondencesTest();
	MultiplicationTest();

	/****************************/
	//ALGORITHM
	/****************************/
	const auto testCloud = LoadCloud("data/rose.obj");
	const auto testCorrupted = LoadCloud("data/rose.obj");

	const auto hostBefore = CommonToThrustVector(testCloud);
	const auto hostAfter = CommonToThrustVector(testCorrupted);

	Cloud deviceCloudBefore = hostBefore;
	Cloud deviceCloudAfter = hostAfter;

	Cloud calculatedCloud(hostAfter.size());

	const auto scaleInput = Functors::ScaleTransform(1000.f);
	thrust::transform(thrust::device, deviceCloudBefore.begin(), deviceCloudBefore.end(), deviceCloudBefore.begin(), scaleInput);
	const auto scaleInputCorrupted = Functors::ScaleTransform(1000.f);
	thrust::transform(thrust::device, deviceCloudAfter.begin(), deviceCloudAfter.end(), deviceCloudAfter.begin(), scaleInputCorrupted);

	const auto sampleTransform = glm::rotate(glm::translate(glm::mat4(1), { 0.05f, 0.05f, 0.05f }), glm::radians(5.f), { 0.5f, 0.5f, 0.5f });
	TransformCloud(deviceCloudAfter, deviceCloudAfter, sampleTransform);

	auto start = std::chrono::high_resolution_clock::now();
	const auto result = CudaICP(deviceCloudBefore, deviceCloudAfter);
	auto stop = std::chrono::high_resolution_clock::now();
	printf("Size: %d points, duration: %dms\n", testCloud.size(), std::chrono::duration_cast<std::chrono::milliseconds>(stop - start));

	TransformCloud(deviceCloudBefore, calculatedCloud, result);

	Common::Renderer renderer(
		Common::ShaderType::SimpleModel,
		ThrustToCommonVector(deviceCloudBefore), //grey
		ThrustToCommonVector(deviceCloudAfter), //blue
		ThrustToCommonVector(calculatedCloud), //red
		std::vector<Point_f>(1));

	renderer.Show();
}
