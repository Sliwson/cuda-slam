#include "hip/hip_runtime.h"
#include "cpdcuda.cuh"
#include "functors.cuh"
#include "svdparams.cuh"
#include "timer.h"
#include "testutils.h"
#include "cudaprobabilities.h"
#include "mstepparams.cuh"
#include "common.h"
#include "cpdutils.h"

using namespace CUDACommon;
using namespace MStepParams;

namespace
{
	typedef thrust::device_vector<glm::vec3> GpuCloud;

	float CalculateSigmaSquared(const GpuCloud& cloudBefore, const GpuCloud& cloudAfter);
	void ComputePMatrix(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudTransformed,
		CUDAProbabilities::Probabilities& probabilities,
		const float& constant,
		const float& sigmaSquared,
		const bool& doTruncate,
		float truncate);
	void ComputePMatrixFast(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudTransformed,
		CUDAProbabilities::Probabilities& probabilities,
		const float& constant,
		const float& weight,
		float* sigmaSquared,
		const float& sigmaSquaredInit,
		const ApproximationType& fgt,
		const std::vector<Point_f>& cloudBeforeCPU,
		const std::vector<Point_f>& cloudAfterCPU,
		const glm::mat3& rotationMatrix,
		const glm::vec3& translationVector,
		const float& scale);
	void ComputePMatrixWithFGTOnCPU(
		const std::vector<Point_f>& cloudBeforeCPU,
		const std::vector<Point_f>& cloudAfterCPU,
		CUDAProbabilities::Probabilities& probabilities,
		const float& weight,
		const float& sigmaSquared,
		const float& sigmaSquaredInit,
		const glm::mat3& rotationMatrix,
		const glm::vec3& translationVector,
		const float& scale);
	void MStep(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudAfter,
		const CUDAProbabilities::Probabilities& probabilities,
		CUDAMStepParams& params,
		const bool& const_scale,
		glm::mat3* rotationMatrix,
		glm::vec3* translationVector,
		float* scale,
		float* sigmaSquared);

	float CalculateSigmaSquared(const GpuCloud& cloudBefore, const GpuCloud& cloudAfter)
	{
		if (cloudBefore.size() > cloudAfter.size())
		{
			const auto functor = Functors::CalculateSigmaSquaredInRow(cloudAfter);
			return thrust::transform_reduce(thrust::device, cloudBefore.begin(), cloudBefore.end(), functor, 0.0f, thrust::plus<float>()) / (float)(3 * cloudBefore.size() * cloudAfter.size());
		}
		else
		{
			const auto functor = Functors::CalculateSigmaSquaredInRow(cloudBefore);
			return thrust::transform_reduce(thrust::device, cloudAfter.begin(), cloudAfter.end(), functor, 0.0f, thrust::plus<float>()) / (float)(3 * cloudBefore.size() * cloudAfter.size());
		}
		return -1.0f;
	}

	void ComputePMatrix(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudTransformed,
		CUDAProbabilities::Probabilities& probabilities,
		const float& constant,
		const float& sigmaSquared,
		const bool& doTruncate,
		float truncate)
	{
		const float multiplier = -0.5f / sigmaSquared;

		thrust::fill(thrust::device, probabilities.p1.begin(), probabilities.p1.end(), 0.0f);
		thrust::fill(thrust::device, probabilities.px.begin(), probabilities.px.end(), glm::vec3(0.0f));

		thrust::counting_iterator<int> idxfirst = thrust::make_counting_iterator<int>(0);
		thrust::counting_iterator<int> idxlast = thrust::make_counting_iterator<int>(cloudTransformed.size());

		auto cloudTransformed_first = thrust::make_zip_iterator<>(thrust::make_tuple(cloudTransformed.begin(), idxfirst));
		auto cloudTransformed_last = thrust::make_zip_iterator(thrust::make_tuple(cloudTransformed.end(), idxlast));

		probabilities.error = 0.0f;
		if (doTruncate)
			truncate = std::log(truncate);

		for (size_t x = 0; x < cloudBefore.size(); x++)
		{
			const auto functorDenominator = Functors::CalculateDenominator(cloudBefore[x], probabilities.p, multiplier, doTruncate, truncate);
			thrust::transform(thrust::device, cloudTransformed_first, cloudTransformed_last, probabilities.tmp.begin(), functorDenominator);
			const float denominator = thrust::reduce(thrust::device, probabilities.tmp.begin(), probabilities.tmp.end(), constant, thrust::plus<float>());
			probabilities.pt1[x] = 1.0f - constant / denominator;

			const auto functor = Functors::CalculateP1AndPX(cloudBefore[x], probabilities.p, probabilities.p1, probabilities.px, denominator);
			thrust::for_each(thrust::device, idxfirst, idxlast, functor);
			probabilities.error -= std::log(denominator);
		}
		probabilities.error += DIMENSION * cloudBefore.size() * std::log(sigmaSquared) / 2.0f;
	}

	void ComputePMatrixFast(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudTransformed,
		CUDAProbabilities::Probabilities& probabilities,
		const float& constant,
		const float& weight,
		float* sigmaSquared,
		const float& sigmaSquaredInit,
		const ApproximationType& fgt,
		const std::vector<Point_f>& cloudBeforeCPU,
		const std::vector<Point_f>& cloudAfterCPU,
		const glm::mat3& rotationMatrix,
		const glm::vec3& translationVector,
		const float& scale)
	{
		if (fgt == ApproximationType::Full)
		{
			if (*sigmaSquared < 0.05)
				*sigmaSquared = 0.05;
			ComputePMatrixWithFGTOnCPU(cloudBeforeCPU, cloudAfterCPU, probabilities, weight, *sigmaSquared, sigmaSquaredInit, rotationMatrix, translationVector, scale);
		}
		else if (fgt == ApproximationType::Hybrid)
		{
			if (*sigmaSquared > 0.015 * sigmaSquaredInit)
				ComputePMatrixWithFGTOnCPU(cloudBeforeCPU, cloudAfterCPU, probabilities, weight, *sigmaSquared, sigmaSquaredInit, rotationMatrix, translationVector, scale);
			else
				ComputePMatrix(cloudBefore, cloudTransformed, probabilities, constant, *sigmaSquared, true, 1e-3f);
		}
	}

	void ComputePMatrixWithFGTOnCPU(
		const std::vector<Point_f>& cloudBeforeCPU,
		const std::vector<Point_f>& cloudAfterCPU,
		CUDAProbabilities::Probabilities& probabilities,
		const float& weight,
		const float& sigmaSquared,
		const float& sigmaSquaredInit,
		const glm::mat3& rotationMatrix,
		const glm::vec3& translationVector,
		const float& scale)
	{
		auto cloudTransformedCPU = Common::GetTransformedCloud(cloudAfterCPU, rotationMatrix, translationVector, scale);
		auto prob = CPDutils::ComputePMatrixWithFGT(cloudBeforeCPU, cloudTransformedCPU, weight, sigmaSquared, sigmaSquaredInit);
		Eigen::Matrix<float, -1, 3, Eigen::RowMajor> px = prob.px;
		hipMemcpy(thrust::raw_pointer_cast(probabilities.p1.data()), prob.p1.data(), cloudAfterCPU.size() * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(probabilities.pt1.data()), prob.pt1.data(), cloudBeforeCPU.size() * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(probabilities.px.data()), px.data(), cloudAfterCPU.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
		probabilities.error = prob.error;
	}

	void MStep(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudAfter,
		const CUDAProbabilities::Probabilities& probabilities,
		CUDAMStepParams& params,
		const bool& const_scale,
		glm::mat3* rotationMatrix,
		glm::vec3* translationVector,
		float* scale,
		float* sigmaSquared)
	{
		const float alpha = 1.f, beta = 0.f;
		const int beforeSize = cloudBefore.size();
		const int afterSize = cloudAfter.size();
		const float Np = thrust::reduce(thrust::device, probabilities.p1.begin(), probabilities.p1.end(), 0.0f, thrust::plus<float>());
		const float InvertedNp = 1.0f / Np;

		//create array beforeT
		auto countingBeforeBegin = thrust::make_counting_iterator<int>(0);
		auto countingBeforeEnd = thrust::make_counting_iterator<int>(beforeSize);
		auto zipBeforeBegin = thrust::make_zip_iterator(thrust::make_tuple(countingBeforeBegin, cloudBefore.begin()));
		auto zipBeforeEnd = thrust::make_zip_iterator(thrust::make_tuple(countingBeforeEnd, cloudBefore.end()));

		auto convertBefore = Functors::GlmToCuBlas(false, beforeSize, params.beforeT);
		thrust::for_each(thrust::device, zipBeforeBegin, zipBeforeEnd, convertBefore);

		//create array afterT
		auto countingAfterBegin = thrust::make_counting_iterator<int>(0);
		auto countingAfterEnd = thrust::make_counting_iterator<int>(afterSize);
		auto zipAfterBegin = thrust::make_zip_iterator(thrust::make_tuple(countingAfterBegin, cloudAfter.begin()));
		auto zipAfterEnd = thrust::make_zip_iterator(thrust::make_tuple(countingAfterEnd, cloudAfter.end()));

		auto convertAfter = Functors::GlmToCuBlas(false, afterSize, params.afterT);
		thrust::for_each(thrust::device, zipAfterBegin, zipAfterEnd, convertAfter);

		//create array px
		auto countingPXBegin = thrust::make_counting_iterator<int>(0);
		auto countingPXEnd = thrust::make_counting_iterator<int>(probabilities.px.size());
		auto zipPXBegin = thrust::make_zip_iterator(thrust::make_tuple(countingPXBegin, probabilities.px.begin()));
		auto zipPXEnd = thrust::make_zip_iterator(thrust::make_tuple(countingPXEnd, probabilities.px.end()));

		auto convertPX = Functors::GlmToCuBlas(true, probabilities.px.size(), params.px);
		thrust::for_each(thrust::device, zipPXBegin, zipPXEnd, convertPX);

		hipblasSgemv(params.multiplyHandle, HIPBLAS_OP_N, 3, beforeSize, &InvertedNp, params.beforeT, 3, params.pt1, 1, &beta, params.centerBefore, 1);

		hipblasSgemv(params.multiplyHandle, HIPBLAS_OP_N, 3, afterSize, &InvertedNp, params.afterT, 3, params.p1, 1, &beta, params.centerAfter, 1);

		hipblasSgemm(params.multiplyHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, afterSize, &alpha, params.afterT, 3, params.px, afterSize, &beta, params.afterTxPX, 3);

		hipblasSgemm(params.multiplyHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 3, 3, 1, &Np, params.centerBefore, 3, params.centerAfter, 3, &beta, params.centerBeforexCenterAfter, 3);

		float minus = -1.0f;
		hipblasSgeam(params.multiplyHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 3, 3, &alpha, params.afterTxPX, 3, &minus, params.centerBeforexCenterAfter, 3, params.AMatrix, 3);

		//TODO: try jacobi svd
		//SVD
		hipsolverDnSgesvd(params.solverHandle, 'A', 'A', 3, 3, params.AMatrix, 3, params.S, params.U, 3, params.VT, 3, params.work, params.workSize, nullptr, params.devInfo);
		int svdResultInfo = 0;
		hipMemcpy(&svdResultInfo, params.devInfo, sizeof(int), hipMemcpyDeviceToHost);
		if (svdResultInfo != 0)
			printf("Svd execution failed!\n");

		float hostS[3], hostVT[9], hostU[9];
		const int copySize = 9 * sizeof(float);
		hipMemcpy(hostS, params.S, 3 * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(hostVT, params.VT, copySize, hipMemcpyDeviceToHost);
		hipMemcpy(hostU, params.U, copySize, hipMemcpyDeviceToHost);

		auto gVT = glm::transpose(CreateGlmMatrix(hostVT));
		auto gU = glm::transpose(CreateGlmMatrix(hostU));

		//revert signs to match svd cpu solution
		for (int i = 0; i < 3; i++)
		{
			gU[0][i] = -gU[0][i];
			gU[1][i] = -gU[1][i];
			gVT[i][0] = -gVT[i][0];
			gVT[i][1] = -gVT[i][1];
		}

		const float determinant = glm::determinant(gU * gVT);
		const auto diagonal = glm::diagonal3x3(glm::vec3{ 1.f, 1.f, determinant });
		*rotationMatrix = gU * diagonal * gVT;
		const auto scaleNumeratorMatrix = glm::diagonal3x3(glm::make_vec3(hostS)) * diagonal;
		const float scaleNumerator = scaleNumeratorMatrix[0][0] + scaleNumeratorMatrix[1][1] + scaleNumeratorMatrix[2][2];

		auto countingSigmaSubtrahendBegin = thrust::make_counting_iterator<int>(0);
		auto countingSigmaSubtrahendEnd = thrust::make_counting_iterator<int>(3 * beforeSize);
		auto zipSigmaSubtrahendBegin = thrust::make_zip_iterator(thrust::make_tuple(countingSigmaSubtrahendBegin, params.beforeT));
		auto zipSigmaSubtrahendEnd = thrust::make_zip_iterator(thrust::make_tuple(countingSigmaSubtrahendEnd, params.beforeT + 3 * beforeSize));

		auto calculateSigmaSubtrahend = Functors::CalculateSigmaSubtrahend(params.pt1);

		thrust::transform(thrust::device, zipSigmaSubtrahendBegin, zipSigmaSubtrahendEnd, params.beforeT, calculateSigmaSubtrahend);
		float sigmaSubtrahend = thrust::reduce(thrust::device, params.beforeT, params.beforeT + 3 * beforeSize, 0.0f, thrust::plus<float>());

		auto countingScaleDenominatorBegin = thrust::make_counting_iterator<int>(0);
		auto countingScaleDenominatorEnd = thrust::make_counting_iterator<int>(3 * afterSize);
		auto zipScaleDenominatorBegin = thrust::make_zip_iterator(thrust::make_tuple(countingScaleDenominatorBegin, params.afterT));
		auto zipScaleDenominatorEnd = thrust::make_zip_iterator(thrust::make_tuple(countingScaleDenominatorEnd, params.afterT + 3 * afterSize));

		auto calculateScaleDenominator = Functors::CalculateSigmaSubtrahend(params.p1);

		thrust::transform(thrust::device, zipScaleDenominatorBegin, zipScaleDenominatorEnd, params.afterT, calculateScaleDenominator);
		float scaleDenominator = thrust::reduce(thrust::device, params.afterT, params.afterT + 3 * afterSize, 0.0f, thrust::plus<float>());

		float hostCenterBefore[3], hostCenterAfter[3];
		hipMemcpy(hostCenterBefore, params.centerBefore, 3 * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(hostCenterAfter, params.centerAfter, 3 * sizeof(float), hipMemcpyDeviceToHost);

		auto glmCenterBefore = glm::make_vec3(hostCenterBefore);
		auto glmCenterAfter = glm::make_vec3(hostCenterAfter);

		sigmaSubtrahend -= Np * glm::dot(glmCenterBefore, glmCenterBefore);
		scaleDenominator -= Np * glm::dot(glmCenterAfter, glmCenterAfter);

		if (const_scale == false)
		{
			*scale = scaleNumerator / scaleDenominator;
			*sigmaSquared = (InvertedNp * std::abs(sigmaSubtrahend - (*scale) * scaleNumerator)) / (float)DIMENSION;
		}
		else
		{
			*sigmaSquared = (InvertedNp * std::abs(sigmaSubtrahend + scaleDenominator - 2 * scaleNumerator)) / (float)DIMENSION;
		}

		*translationVector = glmCenterBefore - (*scale) * (*rotationMatrix) * glmCenterAfter;
	}

	glm::mat4 CudaCPD(
		const GpuCloud& cloudBefore,
		const GpuCloud& cloudAfter,
		int* iterations,
		float* error,
		float eps,
		float weight,
		bool const_scale,
		int maxIterations,
		float tolerance,
		Common::ApproximationType fgt,
		const std::vector<Point_f>& cloudBeforeCPU,
		const std::vector<Point_f>& cloudAfterCPU)
	{
		//allocate memory
		CUDAProbabilities::Probabilities probabilities(cloudBefore.size(), cloudAfter.size());
		CUDAMStepParams mStepParams(cloudBefore.size(), cloudAfter.size(), probabilities);

		*iterations = 0;
		*error = 1e5;
		glm::mat3 rotationMatrix = glm::mat3(1.0f);
		glm::vec3 translationVector = glm::vec3(0.0f);
		float scale = 1.0f;
		float sigmaSquared = CalculateSigmaSquared(cloudBefore, cloudAfter);
		float sigmaSquared_init = sigmaSquared;

		if (weight <= 0.0f)
			weight = 1e-6f;
		if (weight >= 1.0f)
			weight = 1.0f - 1e-6f;

		const float constant = (std::pow(2 * M_PI * sigmaSquared, (float)DIMENSION * 0.5f) * weight * cloudAfter.size()) / ((1 - weight) * cloudBefore.size());
		float ntol = tolerance + 10.0f;
		float l = 0.0f;
		GpuCloud transformedCloud = cloudAfter;
		//EM optimization
		while (*iterations < maxIterations && ntol > tolerance && sigmaSquared > eps)
		{
			//E-step
			if (fgt == Common::ApproximationType::None)
				ComputePMatrix(cloudBefore, transformedCloud, probabilities, constant, sigmaSquared, false, -1.0f);
			else
				ComputePMatrixFast(cloudBefore, transformedCloud, probabilities, constant, weight, &sigmaSquared, sigmaSquared_init, fgt, cloudBeforeCPU, cloudAfterCPU, rotationMatrix, translationVector, scale);

			ntol = std::abs((probabilities.error - l) / probabilities.error);
			l = probabilities.error;

			//M-step
			MStep(cloudBefore, cloudAfter, probabilities, mStepParams, const_scale, &rotationMatrix, &translationVector, &scale, &sigmaSquared);

			TransformCloud(cloudAfter, transformedCloud, ConvertToTransformationMatrix(scale * rotationMatrix, translationVector));
			(*error) = sigmaSquared;
			(*iterations)++;
		}
		mStepParams.Free();
		return ConvertToTransformationMatrix(scale * rotationMatrix, translationVector);
	}
}

void CPDTest()
{
	srand(666);
	const char* objectPath = "data/bunny.obj";
	int pointCount = 5000;
	float testEps = 1e-4f;
	float weight = 0.1f;
	bool const_scale = false;
	const int max_iterations = 50;
	Common::ApproximationType fgt = Common::ApproximationType::Hybrid;

	//reading data from terminal
	//
	/*
	int weight_int = 0;
	int const_scale_int = 0;
	int fgt_int = 0;

	std::cout << "Point count" << std::endl;
	std::cin >> pointCount;
	std::cout << "Weight * 100" << std::endl;
	std::cin >> weight_int;
	std::cout << "const_scale [0-false, 1-true]" << std::endl;
	std::cin >> const_scale_int;
	std::cout << "fgt type [0-None, 1-Hybrid, 2-Full]" << std::endl;
	std::cin >> fgt_int;

	weight = (float)weight_int / 100.0f;

	if (const_scale_int == 0) const_scale = false;
	if (const_scale_int == 1) const_scale = true;

	if (fgt_int == 0) fgt = Common::ApproximationType::None;
	if (fgt_int == 1) fgt = Common::ApproximationType::Hybrid;
	if (fgt_int == 2) fgt = Common::ApproximationType::Full;

	std::cout << "Point count: " << pointCount << " weight: " << weight << " const_scale: " << const_scale << " fgt: " << (int)(fgt) << std::endl;
	//
	*/
	const float scale = 2.0f;

	int iterations = 0;
	float error = 1.0f;
	Timer timer("Cpu timer");

	timer.StartStage("cloud-loading");
	auto cloud = LoadCloud(objectPath);
	timer.StopStage("cloud-loading");
	printf("Cloud size: %d\n", cloud.size());

	timer.StartStage("processing");
	std::transform(cloud.begin(), cloud.end(), cloud.begin(), [](const Point_f& point) { return Point_f{ point.x * 100.f, point.y * 100.f, point.z * 100.f }; });
	if (pointCount > 0)
		cloud.resize(pointCount);

	int cloudSize = cloud.size();
	printf("Processing %d points\n", cloudSize);

	const auto translation_vector = glm::vec3(15.0f, 0.0f, 0.0f);
	const auto rotation_matrix = Tests::GetRotationMatrix({ 1.0f, 0.4f, -0.3f }, glm::radians(50.0f));

	const auto transform = ConvertToTransformationMatrix(scale * rotation_matrix, translation_vector);
	//const auto transform = GetRandomTransformMatrix({ 0.f, 0.f, 0.f }, { 10.0f, 10.0f, 10.0f }, glm::radians(35.f));
	const auto permutation = GetRandomPermutationVector(cloudSize);
	auto permutedCloud = ApplyPermutation(cloud, permutation);
	//std::transform(permutedCloud.begin(), permutedCloud.end(), permutedCloud.begin(), [](const Point_f& point) { return Point_f{ point.x * 2.f, point.y * 2.f, point.z * 2.f }; });
	const auto transformedCloud = GetTransformedCloud(cloud, transform);
	const auto transformedPermutedCloud = GetTransformedCloud(permutedCloud, transform);
	timer.StopStage("processing");

	const auto hostBefore = CommonToThrustVector(transformedPermutedCloud);
	const auto hostAfter = CommonToThrustVector(cloud);

	GpuCloud deviceCloudBefore = hostBefore;
	GpuCloud deviceCloudAfter = hostAfter;

	timer.StartStage("cpd1");
	const auto icpCalculatedTransform1 = CudaCPD(deviceCloudBefore, deviceCloudAfter, &iterations, &error, testEps, weight, const_scale, max_iterations, testEps, fgt, transformedPermutedCloud, cloud);
	timer.StopStage("cpd1");

	printf("ICP test (%d iterations) error = %g\n", iterations, error);

	std::cout << "Transform Matrix" << std::endl;
	PrintMatrix(transform);
	std::cout << "Inverted Transform Matrix" << std::endl;
	PrintMatrix(glm::inverse(transform));

	std::cout << "CPD1 Matrix" << std::endl;
	PrintMatrix(icpCalculatedTransform1);

	timer.PrintResults();

	//std::cout << "Before" << std::endl;
	//PrintVector(deviceCloudBefore);
	//std::cout << "After" << std::endl;
	//PrintVector(deviceCloudAfter);

	Common::Renderer renderer(
		Common::ShaderType::SimpleModel,
		cloud, //red
		transformedPermutedCloud, //green
		GetTransformedCloud(cloud, icpCalculatedTransform1), //yellow
		//GetTransformedCloud(cloud, icpCalculatedTransform2.first, icpCalculatedTransform2.second)); //blue
		std::vector<Point_f>(1)); //green

	renderer.Show();
}
