#include "hip/hip_runtime.h"
#include "cpdcuda.cuh"
#include "functors.cuh"
#include "svdparams.cuh"
#include "timer.h"
#include "testutils.h"
#include "cudaprobabilities.h"
#include "mstepparams.cuh"
#include "common.h"

using namespace CUDACommon;
using namespace CUDAProbabilities;
using namespace MStepParams;

namespace
{
	typedef thrust::device_vector<glm::vec3> Cloud;

	float CalculateSigmaSquared(const Cloud& cloudBefore, const Cloud& cloudAfter);
	void ComputePMatrix(
		const Cloud& cloudBefore,
		const Cloud& cloudTransformed,
		Probabilities& probabilities,
		const float& constant,
		const float& sigmaSquared,
		const bool& doTruncate,
		float truncate);
	void MStep(
		const Cloud& cloudBefore,
		const Cloud& cloudAfter,
		const Probabilities& probabilities,
		CUDAMStepParams& params,
		const bool& const_scale,
		glm::mat3* rotationMatrix,
		glm::vec3* translationVector,
		float* scale,
		float* sigmaSquared);

	float CalculateSigmaSquared(const Cloud& cloudBefore, const Cloud& cloudAfter)
	{
		if (cloudBefore.size() > cloudAfter.size())
		{
			const auto functor = Functors::CalculateSigmaSquaredInRow(cloudAfter);
			return thrust::transform_reduce(thrust::device, cloudBefore.begin(), cloudBefore.end(), functor, 0.0f, thrust::plus<float>()) / (float)(3 * cloudBefore.size() * cloudAfter.size());
		}
		else
		{
			const auto functor = Functors::CalculateSigmaSquaredInRow(cloudBefore);
			return thrust::transform_reduce(thrust::device, cloudAfter.begin(), cloudAfter.end(), functor, 0.0f, thrust::plus<float>()) / (float)(3 * cloudBefore.size() * cloudAfter.size());
		}
		return -1.0f;
	}

	void ComputePMatrix(
		const Cloud& cloudBefore,
		const Cloud& cloudTransformed,
		Probabilities& probabilities,
		const float& constant,
		const float& sigmaSquared,
		const bool& doTruncate,
		float truncate)
	{
		const float multiplier = -0.5f / sigmaSquared;
		/*thrust::device_vector<float> p(cloudTransformed.size());
		thrust::device_vector<float> p1(cloudTransformed.size());
		thrust::device_vector<float> pt1(cloudBefore.size());
		thrust::device_vector<glm::vec3> px(cloudTransformed.size());
		thrust::device_vector<float> tmp(cloudTransformed.size());*/

		thrust::counting_iterator<int> idxfirst = thrust::make_counting_iterator<int>(0);
		thrust::counting_iterator<int> idxlast = thrust::make_counting_iterator<int>(cloudTransformed.size());

		////maybe use auto instead of this
		//thrust::zip_iterator<thrust::tuple<Cloud::iterator, thrust::counting_iterator<int>>> cloudTransformed_first = thrust::make_zip_iterator(thrust::make_tuple(cloudTransformed.begin(), idxfirst));
		auto cloudTransformed_first = thrust::make_zip_iterator<>(thrust::make_tuple(cloudTransformed.begin(), idxfirst));
		auto cloudTransformed_last = thrust::make_zip_iterator(thrust::make_tuple(cloudTransformed.end(), idxlast));
		//thrust::zip_iterator<thrust::tuple<Cloud::iterator, thrust::counting_iterator<int>>> cloudTransformed_last = thrust::make_zip_iterator(thrust::make_tuple(cloudTransformed.end(), idxlast));

		//auto cloudTransformed_first = thrust::make_zip_iterator(thrust::make_tuple(p.begin(), idxfirst));
		//auto cloudTransformed_last = thrust::make_zip_iterator(thrust::make_tuple(p.end(), idxlast));

		probabilities.error = 0.0f;
		if (doTruncate)
			truncate = std::log(truncate);

		for (size_t x = 0; x < cloudBefore.size(); x++)
		{
			const auto functorDenominator = Functors::CalculateDenominator(cloudBefore[x], probabilities.p, multiplier, doTruncate, truncate);
			//const auto functorDenominator = Functors::CalculateDenominator2();
			//const float denominator = thrust::transform_reduce(thrust::device, cloudTransformed_first, cloudTransformed_last, functorDenominator, constant, thrust::plus<float>());
			thrust::transform(thrust::device, cloudTransformed_first, cloudTransformed_last, probabilities.tmp.begin(), functorDenominator);
			const float denominator = thrust::reduce(thrust::device, probabilities.tmp.begin(), probabilities.tmp.end(), constant, thrust::plus<float>());
			//const float denominator = 1;

			std::cout << "denominator: " << denominator << std::endl;

			probabilities.pt1[x] = 1.0f - constant / denominator;

			const auto functor = Functors::CalculateP1AndPX(cloudBefore[x], probabilities.p, probabilities.p1, probabilities.px, denominator);
			thrust::for_each(thrust::device, idxfirst, idxlast, functor);
			probabilities.error -= std::log(denominator);
		}
		probabilities.error += DIMENSION * cloudBefore.size() * std::log(sigmaSquared) / 2.0f;
	}

	void MStep(		
		const Cloud& cloudBefore,
		const Cloud& cloudAfter,
		const Probabilities& probabilities,
		CUDAMStepParams& params,
		const bool& const_scale,
		glm::mat3* rotationMatrix,
		glm::vec3* translationVector,
		float* scale,
		float* sigmaSquared)
	{
		const float alpha = 1.f, beta = 0.f;
		const int beforeSize = cloudBefore.size();
		const int afterSize = cloudAfter.size();
		const float Np = thrust::reduce(thrust::device, probabilities.p1.begin(), probabilities.p1.end(), 0.0f, thrust::plus<float>());
		const float InvertedNp = 1.0f / Np;

		//create array beforeT
		auto countingBeforeBegin = thrust::make_counting_iterator<int>(0);
		auto countingBeforeEnd = thrust::make_counting_iterator<int>(beforeSize);
		auto zipBeforeBegin = thrust::make_zip_iterator(thrust::make_tuple(countingBeforeBegin, cloudBefore.begin()));
		auto zipBeforeEnd = thrust::make_zip_iterator(thrust::make_tuple(countingBeforeEnd, cloudBefore.end()));

		auto convertBefore = Functors::GlmToCuBlas(false, beforeSize, params.beforeT);
		thrust::for_each(thrust::device, zipBeforeBegin, zipBeforeEnd, convertBefore);

		//create array afterT
		auto countingAfterBegin = thrust::make_counting_iterator<int>(0);
		auto countingAfterEnd = thrust::make_counting_iterator<int>(afterSize);
		auto zipAfterBegin = thrust::make_zip_iterator(thrust::make_tuple(countingAfterBegin, cloudAfter.begin()));
		auto zipAfterEnd = thrust::make_zip_iterator(thrust::make_tuple(countingAfterEnd, cloudAfter.end()));

		auto convertAfter = Functors::GlmToCuBlas(false, afterSize, params.afterT);
		thrust::for_each(thrust::device, zipAfterBegin, zipAfterEnd, convertAfter);
		
		//create array px
		auto countingPXBegin = thrust::make_counting_iterator<int>(0);
		auto countingPXEnd = thrust::make_counting_iterator<int>(probabilities.px.size());
		auto zipPXBegin = thrust::make_zip_iterator(thrust::make_tuple(countingPXBegin, probabilities.px.begin()));
		auto zipPXEnd = thrust::make_zip_iterator(thrust::make_tuple(countingPXEnd, probabilities.px.end()));

		auto convertPX = Functors::GlmToCuBlas(true, probabilities.px.size(), params.px);
		thrust::for_each(thrust::device, zipPXBegin, zipPXEnd, convertPX);

		hipblasSgemv(params.multiplyHandle, HIPBLAS_OP_N, 3, beforeSize, &InvertedNp, params.beforeT, 3, params.pt1, 1, &beta, params.centerBefore, 1);

		hipblasSgemv(params.multiplyHandle, HIPBLAS_OP_N, 3, afterSize, &InvertedNp, params.afterT, 3, params.p1, 1, &beta, params.centerAfter, 1);

		hipblasSgemm(params.multiplyHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, afterSize, &alpha, params.afterT, 3, params.px, afterSize, &beta, params.afterTxPX, 3);

		hipblasSgemm(params.multiplyHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 3, 3, 1, &Np, params.centerBefore, 3, params.centerAfter, 3, &beta, params.centerBeforexCenterAfter, 3);

		float minus = -1.0f;
		hipblasSgeam(params.multiplyHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 3, 3, &alpha, params.afterTxPX, 3, &minus, params.centerBeforexCenterAfter, 3, params.AMatrix, 3);

		//TODO: try jacobi svd
		//SVD
		hipsolverDnSgesvd(params.solverHandle, 'A', 'A', 3, 3, params.AMatrix, 3, params.S, params.U, 3, params.VT, 3, params.work, params.workSize, nullptr, params.devInfo);
		int svdResultInfo = 0;
		hipMemcpy(&svdResultInfo, params.devInfo, sizeof(int), hipMemcpyDeviceToHost);
		if (svdResultInfo != 0)
			printf("Svd execution failed!\n");

		float hostS[3], hostVT[9], hostU[9];
		const int copySize = 9 * sizeof(float);
		hipMemcpy(hostS, params.S, 3 * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(hostVT, params.VT, copySize, hipMemcpyDeviceToHost);
		hipMemcpy(hostU, params.U, copySize, hipMemcpyDeviceToHost);

		auto gVT = glm::transpose(CreateGlmMatrix(hostVT));
		auto gU = glm::transpose(CreateGlmMatrix(hostU));

		//revert signs to match svd cpu solution
		for (int i = 0; i < 3; i++)
		{
			gU[0][i] = -gU[0][i];
			gU[1][i] = -gU[1][i];
			gVT[i][0] = -gVT[i][0];
			gVT[i][1] = -gVT[i][1];
		}

		const float determinant = glm::determinant(gU * gVT);
		const auto diagonal = glm::diagonal3x3(glm::vec3{ 1.f, 1.f, determinant });
		*rotationMatrix = gU * diagonal * gVT;
		const auto scaleNumeratorMatrix = glm::diagonal3x3(glm::make_vec3(hostS)) * diagonal;
		const float scaleNumerator = scaleNumeratorMatrix[0][0] + scaleNumeratorMatrix[1][1] + scaleNumeratorMatrix[2][2];



		float BeforeCPU[30];
		hipMemcpy(BeforeCPU, params.beforeT, 30 * sizeof(float), hipMemcpyDeviceToHost);

		float AfterCPU[30];
		hipMemcpy(AfterCPU, params.afterT, 30 * sizeof(float), hipMemcpyDeviceToHost);

		float p1CPU[10];
		hipMemcpy(p1CPU, params.p1, 10 * sizeof(float), hipMemcpyDeviceToHost);

		float pt1CPU[10];
		hipMemcpy(pt1CPU, params.pt1, 10 * sizeof(float), hipMemcpyDeviceToHost);

		float centerBeforeCPU[3];
		hipMemcpy(centerBeforeCPU, params.centerBefore, 3 * sizeof(float), hipMemcpyDeviceToHost);

		float centerAfterCPU[3];
		hipMemcpy(centerAfterCPU, params.centerAfter, 3 * sizeof(float), hipMemcpyDeviceToHost);

		float result[9];
		hipMemcpy(result, params.AMatrix, 9 * sizeof(float), hipMemcpyDeviceToHost);
		
		float afterTxPX[9];
		hipMemcpy(afterTxPX, params.afterTxPX, 9 * sizeof(float), hipMemcpyDeviceToHost);

		float centerBeforexCenterAfter[9];
		hipMemcpy(centerBeforexCenterAfter, params.centerBeforexCenterAfter, 9 * sizeof(float), hipMemcpyDeviceToHost);

		printf("np %f\n", Np);

		printf("BeforeCPU\n");
		for (size_t i = 0; i < 3; i++)
		{
			for (size_t j = 0; j < 10; j++)
			{
				printf("%f ", BeforeCPU[10 * i + j]);
			}
			printf("\n");
		}

		printf("AfterCPU\n");
		for (size_t i = 0; i < 3; i++)
		{
			for (size_t j = 0; j < 10; j++)
			{
				printf("%f ", AfterCPU[10 * i + j]);
			}
			printf("\n");
		}

		printf("p1CPU\n");
		for (size_t j = 0; j < 10; j++)
		{
			printf("%f ", p1CPU[j]);
		}
		printf("\n");

		printf("pt1CPU\n");
		for (size_t j = 0; j < 10; j++)
		{
			printf("%f ", pt1CPU[j]);
		}
		printf("\n");

		printf("centerBeforeCPU\n");
		for (size_t j = 0; j < 3; j++)
		{
			printf("%f ", centerBeforeCPU[j]);
		}
		printf("\n");

		printf("centerAfterCPU\n");
		for (size_t j = 0; j < 3; j++)
		{
			printf("%f ", centerAfterCPU[j]);
		}
		printf("\n");

		printf("afterTxPX\n");
		for (size_t i = 0; i < 3; i++)
		{
			for (size_t j = 0; j < 3; j++)
			{
				printf("%f ", afterTxPX[3 * j + i]);
			}
			printf("\n");
		}

		printf("centerBeforexCenterAfter\n");
		for (size_t i = 0; i < 3; i++)
		{
			for (size_t j = 0; j < 3; j++)
			{
				printf("%f ", centerBeforexCenterAfter[3 * j + i]);
			}
			printf("\n");
		}

		printf("AMatrix\n");
		for (size_t i = 0; i < 3; i++)
		{
			for (size_t j = 0; j < 3; j++)
			{
				printf("%f ", result[3 * j + i]);
			}
			printf("\n");
		}

		printf("matrix U\n");
		Common::PrintMatrix(gU);
		printf("matrix VT\n");
		Common::PrintMatrix(gVT);

		printf("S Matrix\n");
		for (size_t i = 0; i < 3; i++)
		{
			printf("%f ", hostS[i]);
			printf("\n");
		}

		printf("scale numerator %f\n", scaleNumerator);

		/*const Eigen::JacobiSVD<Eigen::MatrixXf> svd = Eigen::JacobiSVD<Eigen::MatrixXf>(AMatrix, Eigen::ComputeThinU | Eigen::ComputeThinV);

		const Eigen::Matrix3f matrixU = svd.matrixU();
		const Eigen::Matrix3f matrixV = svd.matrixV();
		const Eigen::Matrix3f matrixVT = matrixV.transpose();

		const Eigen::Matrix3f determinantMatrix = matrixU * matrixVT;

		const Eigen::Matrix3f diag = Eigen::DiagonalMatrix<float, 3>(1.0f, 1.0f, determinantMatrix.determinant());

		const Eigen::Matrix3f EigenRotationMatrix = matrixU * diag * matrixVT;

		const Eigen::Matrix3f EigenScaleNumerator = svd.singularValues().asDiagonal() * diag;

		const float scaleNumerator = EigenScaleNumerator.trace();
		const float sigmaSubtrahend = (EigenBeforeT.transpose().array().pow(2) * probabilities.pt1.replicate(1, DIMENSION).array()).sum()
			- Np * EigenCenterBefore.transpose() * EigenCenterBefore;
		const float scaleDenominator = (EigenAfterT.transpose().array().pow(2) * probabilities.p1.replicate(1, DIMENSION).array()).sum()
			- Np * EigenCenterAfter.transpose() * EigenCenterAfter;

		if (const_scale == false)
		{
			*scale = scaleNumerator / scaleDenominator;
			*sigmaSquared = (InvertedNp * std::abs(sigmaSubtrahend - (*scale) * scaleNumerator)) / (float)DIMENSION;
		}
		else
		{
			*sigmaSquared = (InvertedNp * std::abs(sigmaSubtrahend + scaleDenominator - 2 * scaleNumerator)) / (float)DIMENSION;
		}

		const Eigen::Vector3f EigenTranslationVector = EigenCenterBefore - (*scale) * EigenRotationMatrix * EigenCenterAfter;

		*translationVector = ConvertTranslationVector(EigenTranslationVector);

		*rotationMatrix = ConvertRotationMatrix(EigenRotationMatrix);*/
	}

	glm::mat4 CudaCPD(
		const Cloud& cloudBefore,
		const Cloud& cloudAfter,
		int* iterations,
		float* error,
		float eps,
		float weight,
		bool const_scale,
		int maxIterations,
		float tolerance,
		Common::ApproximationType fgt)
	{
		*iterations = 0;
		*error = 1e5;
		glm::mat3 rotationMatrix = glm::mat3(1.0f);
		glm::vec3 translationVector = glm::vec3(0.0f);
		float scale = 1.0f;
		float sigmaSquared = CalculateSigmaSquared(cloudBefore, cloudAfter);
		float sigmaSquared_init = sigmaSquared;

		Probabilities probabilities(cloudBefore.size(), cloudAfter.size());
		CUDAMStepParams mStepParams(cloudBefore.size(), cloudAfter.size(), probabilities);

		if (weight <= 0.0f)
			weight = 1e-6f;
		if (weight >= 1.0f)
			weight = 1.0f - 1e-6f;

		const float constant = (std::pow(2 * M_PI * sigmaSquared, (float)DIMENSION * 0.5f) * weight * cloudAfter.size()) / ((1 - weight) * cloudBefore.size());
		float ntol = tolerance + 10.0f;
		float l = 0.0f;
		Cloud transformedCloud = cloudAfter;
		//EM optimization
		while (*iterations < maxIterations && ntol > tolerance && sigmaSquared > eps)
		{
			//E-step
			if (fgt == Common::ApproximationType::None)
				ComputePMatrix(cloudBefore, transformedCloud, probabilities, constant, sigmaSquared, false, -1.0f);
			//else
			//	probabilities = ComputePMatrixFast(cloudBefore, transformedCloud, constant, weight, &sigmaSquared, sigmaSquared_init, fgt);

			ntol = std::abs((probabilities.error - l) / probabilities.error);
			l = probabilities.error;

			thrust::host_vector<float> p1 = probabilities.p1;
			thrust::host_vector<float> pt1 = probabilities.pt1;
			thrust::host_vector<glm::vec3> px = probabilities.px;

			std::cout << "P1" << std::endl;
			PrintVector(probabilities.p1);
			std::cout << std::endl << "Pt1" << std::endl;
			PrintVector(probabilities.pt1);
			std::cout << std::endl << "PX" << std::endl;
			PrintVector(probabilities.px);

			//M-step
			MStep(cloudBefore, cloudAfter, probabilities, mStepParams, const_scale, &rotationMatrix, &translationVector, &scale, &sigmaSquared);

			//transformedCloud = GetTransformedCloud(cloudAfter, rotationMatrix, translationVector, scale);
			(*error) = sigmaSquared;
			(*iterations)++;
			break;
		}
		//return std::make_pair(scale * rotationMatrix, translationVector);

		return glm::mat4(0.0f);
	}
}

void CPDTest()
{
	const char* objectPath = "data/bunny.obj";
	const int pointCount = 10;
	const float testEps = 1e-6f;
	const float weight = 0.0f;
	const bool const_scale = false;
	const int max_iterations = 50;
	const Common::ApproximationType fgt = Common::ApproximationType::None;

	srand(666);
	int iterations = 0;
	float error = 1.0f;
	Timer timer("Cpu timer");

	timer.StartStage("cloud-loading");
	auto cloud = LoadCloud(objectPath);
	timer.StopStage("cloud-loading");
	printf("Cloud size: %d\n", cloud.size());

	timer.StartStage("processing");
	std::transform(cloud.begin(), cloud.end(), cloud.begin(), [](const Point_f& point) { return Point_f{ point.x * 100.f, point.y * 100.f, point.z * 100.f }; });
	if (pointCount > 0)
		cloud.resize(pointCount);

	int cloudSize = cloud.size();
	printf("Processing %d points\n", cloudSize);

	const auto translation_vector = glm::vec3(15.0f, 0.0f, 0.0f);
	const auto rotation_matrix = Tests::GetRotationMatrix({ 1.0f, 0.4f, -0.3f }, glm::radians(50.0f));

	const auto transform = ConvertToTransformationMatrix(rotation_matrix, translation_vector);
	//const auto transform = GetRandomTransformMatrix({ 0.f, 0.f, 0.f }, { 10.0f, 10.0f, 10.0f }, glm::radians(35.f));
	const auto permutation = GetRandomPermutationVector(cloudSize);
	auto permutedCloud = cloud;// ApplyPermutation(cloud, permutation);
	std::transform(permutedCloud.begin(), permutedCloud.end(), permutedCloud.begin(), [](const Point_f& point) { return Point_f{ point.x * 2.f, point.y * 2.f, point.z * 2.f }; });
	const auto transformedCloud = GetTransformedCloud(cloud, transform);
	const auto transformedPermutedCloud = GetTransformedCloud(permutedCloud, transform);
	timer.StopStage("processing");

	const auto hostBefore = CommonToThrustVector(transformedPermutedCloud);
	const auto hostAfter = CommonToThrustVector(cloud);

	Cloud deviceCloudBefore = hostBefore;
	Cloud deviceCloudAfter = hostAfter;

	timer.StartStage("cpd1");
	const auto icpCalculatedTransform1 = CudaCPD(deviceCloudBefore, deviceCloudAfter, &iterations, &error, testEps, weight, const_scale, max_iterations, testEps, fgt);
	timer.StopStage("cpd1");

	//iterations = 0;
	//error = 1.0f;
	//timer.StartStage("icp2");
	////const auto icpCalculatedTransform2 = CoherentPointDrift::GetRigidCPDTransformationMatrix(cloud, transformedPermutedCloud, &iterations, &error, testEps, weigth, const_scale, max_iterations, testEps, fgt);
	//timer.StopStage("icp2");

	//printf("ICP test (%d iterations) error = %g\n", iterations, error);

	std::cout << "Transform Matrix" << std::endl;
	PrintMatrix(transform);
	//std::cout << "Inverted Transform Matrix" << std::endl;
	//PrintMatrix(glm::inverse(transform));

	//std::cout << "CPD1 Matrix" << std::endl;
	//PrintMatrix(icpCalculatedTransform1.first, icpCalculatedTransform1.second);

	//timer.PrintResults();

	std::cout << "Before" << std::endl;
	PrintVector(deviceCloudBefore);
	std::cout << "After" << std::endl;
	PrintVector(deviceCloudAfter);

	//Common::Renderer renderer(
	//	Common::ShaderType::SimpleModel,
	//	cloud, //red
	//	transformedPermutedCloud, //green
	//	GetTransformedCloud(cloud, icpCalculatedTransform1.first, icpCalculatedTransform1.second), //yellow
	//	//GetTransformedCloud(cloud, icpCalculatedTransform2.first, icpCalculatedTransform2.second)); //blue
	//	std::vector<Point_f>(1)); //green

	//renderer.Show();
}