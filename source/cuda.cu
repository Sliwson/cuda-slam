#include "cuda.cuh"
#include "functors.cuh"
#include "svdparams.cuh"

namespace CUDACommon
{
	void PrintVector(thrust::host_vector<float> vector)
	{
		for (int i = 0; i < vector.size(); i++)
		{
			printf("%f\n", vector[i]);
		}
	}

	void PrintVector(thrust::host_vector<glm::vec3> vector)
	{
		for (int i = 0; i < vector.size(); i++)
		{
			printf("%f %f %f\n", vector[i].x, vector[i].y, vector[i].z);
		}
	}

	void PrintVector(thrust::device_vector<float> vector)
	{
		thrust::host_vector<float> vec = vector;
		PrintVector(vec);
	}

	void PrintVector(thrust::device_vector<glm::vec3> vector)
	{
		thrust::host_vector<glm::vec3> vec = vector;
		PrintVector(vec);
	}

	thrust::host_vector<glm::vec3> CommonToThrustVector(const std::vector<Common::Point_f>& vec)
	{
		thrust::host_vector<glm::vec3> hostCloud(vec.size());
		for (int i = 0; i < vec.size(); i++)
			hostCloud[i] = (glm::vec3)vec[i];

		return hostCloud;
	}

	std::vector<Point_f> ThrustToCommonVector(const Cloud& vec)
	{
		thrust::host_vector<glm::vec3> hostCloud = vec;
		std::vector<Point_f> outVector(vec.size());
		for (int i = 0; i < hostCloud.size(); i++)
			outVector[i] = { hostCloud[i].x, hostCloud[i].y, hostCloud[i].z };

		return outVector;
	}

	glm::vec3 CalculateCentroid(const Cloud& vec)
	{
		const auto sum = thrust::reduce(thrust::device, vec.begin(), vec.end());
		return sum / static_cast<float>(vec.size());
	}

	void TransformCloud(const Cloud& vec, Cloud& out, const glm::mat4& transform)
	{
		const auto functor = Functors::MatrixTransform(transform);
		thrust::transform(thrust::device, vec.begin(), vec.end(), out.begin(), functor);
	}

	__device__ float GetDistanceSquared(const glm::vec3& first, const glm::vec3& second)
	{
		const auto d = second - first;
		return d.x * d.x + d.y * d.y + d.z * d.z;
	}

	float GetMeanSquaredError(const IndexIterator& permutation, const Cloud& before, const Cloud& after)
	{
		auto permutationIteratorBegin = thrust::make_permutation_iterator(after.begin(), permutation.begin());
		auto permutationIteratorEnd = thrust::make_permutation_iterator(after.end(), permutation.end());
		auto zipBegin = thrust::make_zip_iterator(thrust::make_tuple(permutationIteratorBegin, before.begin()));
		auto zipEnd = thrust::make_zip_iterator(thrust::make_tuple(permutationIteratorEnd, before.end()));
		auto mseFunctor = Functors::MSETransform();
		auto sumFunctor = thrust::plus<float>();
		auto result = thrust::transform_reduce(thrust::device, zipBegin, zipEnd, mseFunctor, 0.f, sumFunctor);
		return result / after.size();
	}

	void GetAlignedCloud(const Cloud& source, Cloud& target)
	{
		const auto centroid = CalculateCentroid(source);
		const auto transform = Functors::TranslateTransform(-centroid);
		thrust::transform(thrust::device, source.begin(), source.end(), target.begin(), transform);
	}

	void CuBlasMultiply(float* A, float* B, float* C, int size, CudaSvdParams& params)
	{
		const float alpha = 1.f, beta = 0.f;
		hipblasSgemm(params.multiplyHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, size, &alpha, A, 3, B, size, &beta, C, 3);
	}

	glm::mat3 CreateGlmMatrix(float* squareMatrix)
	{
		return glm::transpose(glm::make_mat3(squareMatrix));
	}

	glm::mat4 LeastSquaresSVD(const IndexIterator& permutation, const Cloud& before, const Cloud& after, Cloud& alignBefore, Cloud& alignAfter, CudaSvdParams params)
	{
		const int size = before.size();

		//align arrays
		const auto centroidBefore = CalculateCentroid(before);
		GetAlignedCloud(before, alignBefore);

		auto permutationIteratorBegin = thrust::make_permutation_iterator(after.begin(), permutation.begin());
		auto permutationIteratorEnd = thrust::make_permutation_iterator(after.end(), permutation.end());
		thrust::copy(thrust::device, permutationIteratorBegin, permutationIteratorEnd, alignAfter.begin());
		const auto centroidAfter = CalculateCentroid(alignAfter);
		GetAlignedCloud(alignAfter, alignAfter);

		//create array AFTER (transposed)
		auto countingBegin = thrust::make_counting_iterator<int>(0);
		auto countingEnd = thrust::make_counting_iterator<int>(alignAfter.size());
		auto zipBegin = thrust::make_zip_iterator(thrust::make_tuple(countingBegin, alignAfter.begin()));
		auto zipEnd = thrust::make_zip_iterator(thrust::make_tuple(countingEnd, alignAfter.end()));

		auto convertAfter = Functors::GlmToCuBlas(true, size, params.workAfter);
		thrust::for_each(thrust::device, zipBegin, zipEnd, convertAfter);

		//create array BEFORE
		const auto beforeZipBegin = thrust::make_zip_iterator(thrust::make_tuple(countingBegin, alignBefore.begin()));
		const auto beforeZipEnd = thrust::make_zip_iterator(thrust::make_tuple(countingEnd, alignBefore.end()));
		auto convertBefore = Functors::GlmToCuBlas(false, before.size(), params.workBefore);
		thrust::for_each(thrust::device, beforeZipBegin, beforeZipEnd, convertBefore);

		//multiply
		CuBlasMultiply(params.workBefore, params.workAfter, params.multiplyResult, size, params);
		float result[9];
		hipMemcpy(result, params.multiplyResult, 9 * sizeof(float), hipMemcpyDeviceToHost);
		auto matrix = CreateGlmMatrix(result);
		//return Common::GetTransform(matrix, centroidBefore, centroidAfter);

		float transposed[9];
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				transposed[3 * i + j] = result[3 * j + i];
		hipMemcpy(params.multiplyResult, transposed, 9 * sizeof(float), hipMemcpyHostToDevice);

		//svd
		hipsolverDnSgesvd(params.solverHandle, 'A', 'A', 3, 3, params.multiplyResult, 3, params.S, params.U, 3, params.VT, 3, params.work, params.workSize, nullptr, params.devInfo);
		int svdResultInfo = 0;
		hipMemcpy(&svdResultInfo, params.devInfo, sizeof(int), hipMemcpyDeviceToHost);
		if (svdResultInfo != 0)
			printf("Svd execution failed!\n");

		float hostS[9], hostVT[9], hostU[9];
		const int copySize = 9 * sizeof(float);
		hipMemcpy(hostS, params.S, copySize, hipMemcpyDeviceToHost);
		hipMemcpy(hostVT, params.VT, copySize, hipMemcpyDeviceToHost);
		hipMemcpy(hostU, params.U, copySize, hipMemcpyDeviceToHost);

		auto gVT = glm::transpose(CreateGlmMatrix(hostVT));
		auto gU = glm::transpose(CreateGlmMatrix(hostU));

		//revert signs to match svd cpu solution
		for (int i = 0; i < 3; i++)
		{
			gU[1][i] = -gU[1][i];
			gVT[i][1] = -gVT[i][1];
		}

		const float determinant = glm::determinant(gU * gVT);
		const auto diagonal = glm::diagonal3x3(glm::vec3{ 1.f, 1.f, determinant });
		const auto rotation = gU * diagonal * gVT;

		const auto translation = centroidAfter - rotation * centroidBefore;

		auto transformation = glm::mat4(0.f);
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				transformation[i][j] = rotation[i][j];

		transformation[3][0] = translation.x;
		transformation[3][1] = translation.y;
		transformation[3][2] = translation.z;
		transformation[3][3] = 1.0f;

		return transformation;
	}
}
