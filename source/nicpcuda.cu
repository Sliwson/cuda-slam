#include "hip/hip_runtime.h"
#include "icpcuda.cuh"
#include "functors.cuh"
#include "parallelsvdhelper.cuh"

using namespace CUDACommon;

namespace
{
	void PrepareMatricesForParallelSVD(const GpuCloud& cloudBefore, const GpuCloud& cloudAfter, int batchSize, thrust::host_vector<float*>& outputBefore, thrust::host_vector<float*>& outputAfter)
	{
		int cloudSize = std::min(cloudBefore.size(), cloudAfter.size());

		outputBefore.resize(batchSize);
		outputAfter.resize(batchSize);
		for (int i = 0; i < batchSize; i++)
		{
			hipMalloc(&(outputBefore[i]), 3 * cloudBefore.size() * sizeof(float));
			hipMalloc(&(outputAfter[i]), 3 * cloudAfter.size() * sizeof(float));
		}

		GpuCloud alignBefore(cloudBefore.size());
		GpuCloud alignAfter(cloudAfter.size());

		// Align array
		GetAlignedCloud(cloudBefore, alignBefore);
		GetAlignedCloud(cloudAfter, alignAfter);

		for (int i = 0; i < batchSize; i++)
		{
			// Generate permutation
			std::vector<int> h_permutation = GetRandomPermutationVector(cloudSize);
			IndexIterator d_permutation(h_permutation.size());
			thrust::copy(h_permutation.begin(), h_permutation.end(), d_permutation.begin());
			auto permutedBefore = ApplyPermutation(alignBefore, d_permutation);
			auto permutedAfter = ApplyPermutation(alignAfter, d_permutation);

			// Create counting iterators
			auto beforeCountingBegin = thrust::make_counting_iterator<int>(0);
			auto beforeCountingEnd = thrust::make_counting_iterator<int>(permutedBefore.size());
			auto afterCountingBegin = thrust::make_counting_iterator<int>(0);
			auto afterCountingEnd = thrust::make_counting_iterator<int>(permutedAfter.size());

			// Create array for SVD
			const auto beforeZipBegin = thrust::make_zip_iterator(thrust::make_tuple(beforeCountingBegin, permutedBefore.begin()));
			const auto beforeZipEnd = thrust::make_zip_iterator(thrust::make_tuple(beforeCountingEnd, permutedBefore.end()));
			auto convertBefore = Functors::GlmToCuBlas(true, permutedBefore.size(), outputBefore[i]);
			thrust::for_each(thrust::device, beforeZipBegin, beforeZipEnd, convertBefore);
			const auto afterZipBegin = thrust::make_zip_iterator(thrust::make_tuple(afterCountingBegin, permutedAfter.begin()));
			const auto afterZipEnd = thrust::make_zip_iterator(thrust::make_tuple(afterCountingEnd, permutedAfter.end()));
			auto convertAfter = Functors::GlmToCuBlas(true, permutedAfter.size(), outputAfter[i]);
			thrust::for_each(thrust::device, afterZipBegin, afterZipEnd, convertAfter);
		}
	}

	void GetSVDResultParallel(const GpuCloud& cloudBefore, const GpuCloud& cloudAfter, int batchSize, thrust::host_vector<glm::mat3>& outputBefore, thrust::host_vector<glm::mat3>& outputAfter)
	{
		thrust::host_vector<float*> preparedBefore;
		thrust::host_vector<float*> preparedAfter;

		PrepareMatricesForParallelSVD(cloudBefore, cloudAfter, batchSize, preparedBefore, preparedAfter);

		// Run SVD for cloud before
		CudaParallelSvdHelper svdBefore(batchSize, cloudBefore.size(), 3, false);
		svdBefore.RunSVD(preparedBefore);
		outputBefore = svdBefore.GetHostMatricesVT();
		svdBefore.FreeMemory();

		// Run SVD for cloud after
		CudaParallelSvdHelper svdAfter(batchSize, cloudAfter.size(), 3, false);
		svdAfter.RunSVD(preparedAfter);
		outputAfter = svdAfter.GetHostMatricesVT();
		svdAfter.FreeMemory();

		for (int i = 0; i < batchSize; i++)
		{
			if (preparedBefore[i])
				hipFree(preparedBefore[i]);

			if (preparedAfter[i])
				hipFree(preparedAfter[i]);
		}
	}

	GpuCloud GetSubcloud(const GpuCloud& cloud, int subcloudSize)
	{
		if (subcloudSize >= cloud.size())
			return cloud;
		std::vector<int> subcloudIndices = GetRandomPermutationVector(cloud.size());
		subcloudIndices.resize(subcloudSize);
		thrust::device_vector<int> indices(subcloudIndices);

		GpuCloud subcloud(subcloudIndices.size());
		const auto getSubcloudFunctor = Functors::Permutation(cloud);
		thrust::transform(thrust::device, indices.begin(), indices.end(), subcloud.begin(), getSubcloudFunctor);

		return subcloud;
	}

	glm::mat4 CudaNonIterative(const GpuCloud& before, const GpuCloud& after, int* repetitions, float* error, float eps, int maxRepetitions, int batchSize, const int subcloudSize)
	{
		glm::mat4 transformResult(1.0f);
		*error = std::numeric_limits<float>::max();

		thrust::host_vector<glm::mat3> matricesBefore(batchSize);
		thrust::host_vector<glm::mat3> matricesAfter(batchSize);

		const auto subcloud = GetSubcloud(before, subcloudSize);
		auto batchesCount = maxRepetitions / batchSize;
		auto lastBatchSize = maxRepetitions % batchSize;
		auto threadsToRun = batchSize;

		auto centroidBefore = CalculateCentroid(before);
		auto centroidAfter = CalculateCentroid(after);

		for (int i = 0; i <= batchesCount; i++)
		{
			if (i == batchesCount)
			{
				if (lastBatchSize != 0)
					threadsToRun = lastBatchSize;
				else
					break;
			}

			GetSVDResultParallel(before, after, threadsToRun, matricesBefore, matricesAfter);
			*repetitions += threadsToRun;

			for (int j = 0; j < threadsToRun; j++)
			{
				auto transformationMatrix = glm::mat4(1.0f);
				auto rotationMatrix = matricesAfter[j] * glm::transpose(matricesBefore[j]);
				auto translationVector = centroidAfter - (rotationMatrix * centroidBefore);

				for (int x = 0; x < 3; x++)
					for (int y = 0; y < 3; y++)
						transformationMatrix[x][y] = rotationMatrix[x][y];

				transformationMatrix[3][0] = translationVector.x;
				transformationMatrix[3][1] = translationVector.y;
				transformationMatrix[3][2] = translationVector.z;
				transformationMatrix[3][3] = 1.0f;

				GpuCloud workingSubcloud(subcloud.size());
				thrust::device_vector<int> indices(workingSubcloud.size());
				TransformCloud(subcloud, workingSubcloud, transformationMatrix);
				GetCorrespondingPoints(indices, workingSubcloud, after);
				auto currentError = GetMeanSquaredError(indices, workingSubcloud, after);
				printf("Current error: %f\n", currentError);

				// Process the results
				if (currentError <= eps)
				{
					*error = currentError;
					return transformationMatrix;
				}

				if (currentError < *error)
				{
					*error = currentError;
					transformResult = transformationMatrix;
				}
			}
		}

		return transformResult;
	}
}

void NonIterativeCudaTest()
{
	/****************************/
	//TESTS
	/****************************/
	//MultiplicationTest();

	/****************************/
	//ALGORITHM
	/****************************/
	const auto testCloud = LoadCloud("data/bunny.obj");
	const auto testCorrupted = LoadCloud("data/bunny.obj");
	int repetitions;
	float error;
	const int maxRepetitions = 20;
	const int subcloudSize = 1000;
	const float eps = 1e-5;
	const int cpuThreadsCount = (int)std::thread::hardware_concurrency();
	//testCloud.resize(10000);
	//testCorrupted.resize(10000);

	const auto hostBefore = CommonToThrustVector(testCloud);
	const auto hostAfter = CommonToThrustVector(testCorrupted);

	GpuCloud deviceCloudBefore = hostBefore;
	GpuCloud deviceCloudAfter = hostAfter;

	GpuCloud calculatedCloud(hostAfter.size());

	const auto scaleInput = Functors::ScaleTransform(1000.f);
	thrust::transform(thrust::device, deviceCloudBefore.begin(), deviceCloudBefore.end(), deviceCloudBefore.begin(), scaleInput);
	const auto scaleInputCorrupted = Functors::ScaleTransform(1000.f);
	thrust::transform(thrust::device, deviceCloudAfter.begin(), deviceCloudAfter.end(), deviceCloudAfter.begin(), scaleInputCorrupted);

	const auto sampleTransform = glm::rotate(glm::translate(glm::mat4(1), { 0.05f, 0.05f, 0.05f }), glm::radians(5.f), { 0.5f, 0.5f, 0.5f });
	// TODO: Remove this debug print for sample transformation
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			printf("%f\t", sampleTransform[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	TransformCloud(deviceCloudAfter, deviceCloudAfter, sampleTransform);

	auto start = std::chrono::high_resolution_clock::now();
	const auto result = CudaNonIterative(deviceCloudBefore, deviceCloudAfter, &repetitions, &error, eps, maxRepetitions, cpuThreadsCount, subcloudSize);
	auto stop = std::chrono::high_resolution_clock::now();
	printf("Size: %d points, duration: %dms\n", testCloud.size(), std::chrono::duration_cast<std::chrono::milliseconds>(stop - start));

	TransformCloud(deviceCloudBefore, calculatedCloud, result);

	Common::Renderer renderer(
		Common::ShaderType::SimpleModel,
		ThrustToCommonVector(deviceCloudBefore), //red
		ThrustToCommonVector(deviceCloudAfter), //green
		ThrustToCommonVector(calculatedCloud), //yellow
		std::vector<Point_f>(1));

	renderer.Show();
}
