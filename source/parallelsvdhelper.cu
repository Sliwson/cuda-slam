#include "parallelsvdhelper.cuh"

CudaParallelSvdHelper::CudaParallelSvdHelper(int batchSize, int m, int n, bool useMatrixU, bool useMatrixV)
	:batchSize(batchSize), m(m), n(n), useMatrixU(useMatrixU), useMatrixV(useMatrixV)
{
	info.resize(batchSize);
	for (int i = 0; i < batchSize; i++)
	{
		error = hipMalloc(&(info[i]), sizeof(int));
		assert(error == hipSuccess);
	}

	S.resize(batchSize);
	for (int i = 0; i < batchSize; i++)
	{
		error = hipMalloc(&(S[i]), n * n * sizeof(float));
		assert(error == hipSuccess);
	}

	VT.resize(batchSize);
	if (useMatrixV)
	{
		for (int i = 0; i < batchSize; i++)
		{
			error = hipMalloc(&(VT[i]), n * n * sizeof(float));
			assert(error == hipSuccess);
		}
	}

	U.resize(batchSize);
	if (useMatrixU)
	{
		for (int i = 0; i < batchSize; i++)
		{
			error = hipMalloc(&(U[i]), m * m * sizeof(float));
			assert(error == hipSuccess);
		}
	}

	// Create handles and streams
	solverHandles.resize(batchSize);
	streams.resize(batchSize);
	for (int i = 0; i < batchSize; i++)
	{
		cusolverStatus = hipsolverDnCreate(&(solverHandles[i]));
		assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

		error = hipStreamCreateWithFlags(&(streams[i]), hipStreamNonBlocking);
		assert(error == hipSuccess);

		cusolverStatus = hipsolverSetStream(solverHandles[i], streams[i]);
		assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);
	}

	// Allocate memory for SVD work
	work.resize(batchSize);
	workSize.resize(batchSize);
	for (int i = 0; i < batchSize; i++)
	{
		cusolverStatus = hipsolverDnSgesvd_bufferSize(solverHandles[i], m, n, &(workSize[i]));
		assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

		error = hipMalloc(&(work[i]), workSize[i] * sizeof(float));
		assert(error == hipSuccess);
	}
}

void CudaParallelSvdHelper::RunSVD(const thrust::host_vector<float*>& sourceMatrices)
{
	const auto thread_work = [&](int index) {
		auto status = hipsolverDnSgesvd(solverHandles[index], 'N', 'A', m, n, sourceMatrices[index], m, S[index], U[index], m, VT[index], n, work[index], workSize[index], nullptr, info[index]);
		assert(status == HIPSOLVER_STATUS_SUCCESS);
	};

	std::vector<std::thread> workerThreads(batchSize);

	// SVD needs to be launched from separated threads to take full advantage of CUDA streams
	for (int j = 0; j < batchSize; j++)
		workerThreads[j] = std::thread(thread_work, j);

	// Wait for threads to finish
	for (int j = 0; j < batchSize; j++)
		workerThreads[j].join();

	error = hipDeviceSynchronize();
	assert(error == hipSuccess);
}

thrust::host_vector<glm::mat3> CudaParallelSvdHelper::GetHostMatricesVT()
{
	thrust::host_vector<glm::mat3> result(batchSize);

	float* data = (float*)malloc(n * n * sizeof(float));
	for (int i = 0; i < batchSize; i++)
	{
		// Use V^T matrix instead of U as we pass transposed matrix to cusolver
		// A = U * S * V => A^T = V^T * S^T * U^T => U(A^T)  = V^T (more or less :) )
		error = hipMemcpy(data, VT[i], 9 * sizeof(float), hipMemcpyDeviceToHost);
		assert(error == hipSuccess);

		result[i] = CUDACommon::CreateGlmMatrix(data);
	}
	free(data);

	return result;
}

void CudaParallelSvdHelper::FreeMemory()
{
	for (int i = 0; i < batchSize; i++)
	{
		if (streams[i])
			hipStreamDestroy(streams[i]);

		if (solverHandles[i])
			hipsolverDnDestroy(solverHandles[i]);

		if (work[i])
			hipFree(work[i]);

		if (S[i])
			hipFree(S[i]);

		if (VT[i])
			hipFree(VT[i]);

		if (U[i])
			hipFree(U[i]);

		if (info[i])
			hipFree(info[i]);
	}
}